#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <dragon.h>

#define CUDA_CALL_SAFE(f) \
    do \
    {                                                        \
        hipError_t _cuda_error = f;                         \
        if (_cuda_error != hipSuccess)                      \
        {                                                    \
            fprintf(stderr,  \
                "%s, %d, CUDA ERROR: %s %s\n",  \
                __FILE__,   \
                __LINE__,   \
                hipGetErrorName(_cuda_error),  \
                hipGetErrorString(_cuda_error) \
            ); \
            abort(); \
            return EXIT_FAILURE; \
        } \
    } while (0)        

double time_diff(struct timeval tv_start, struct timeval tv_stop)
{
    return (double)(tv_stop.tv_sec - tv_start.tv_sec) * 1000.0 + (double)(tv_stop.tv_usec - tv_start.tv_usec) / 1000.0;
}

#define NUM_THREADS_PER_BLOCK 1024
#define GPU_PAGE_SIZE (1LLU << 21)

/** 
 * Each block reads 2MB (1 GPU page). As we have 1024 threads per block, each thread reads 2KB.
 */

__global__ void kernel(volatile uint32_t *g_buf) 
{
    int block_id = blockIdx.x;

    // Calculate the start address of this block
    volatile uint32_t *buf = (volatile uint32_t *)(((char *)g_buf) + block_id * GPU_PAGE_SIZE);

    int tid = threadIdx.x;

    int num_elements = GPU_PAGE_SIZE / NUM_THREADS_PER_BLOCK / sizeof(uint32_t);

    uint64_t start, stop;
    uint32_t tmp;

    asm volatile ("mov.u64 %0, %%globaltimer;" : "=l"(start));
    for (int i = 0; i < num_elements; ++i)
        tmp = buf[i * NUM_THREADS_PER_BLOCK + tid];
    asm volatile ("mov.u64 %0, %%globaltimer;" : "=l"(stop));
}

int main(int argc, char *argv[])
{
    volatile uint32_t *g_buf;
    size_t num_tblocks;          
    int size_order;
    size_t total_size;


    float kernel_time = 0;        // in ms
    double free_time = 0;         // in ms
    double map_time = 0;          // in ms
    double bw = 0;                // in MB/s

    hipEvent_t start_event, stop_event;
    struct timeval tv_start, tv_stop;

    int use_direct = 0;
    unsigned int flags = D_F_READ;

    if (argc != 4)
    {
        fprintf(stderr, "Usage: %s <file> <size_in_GiB> <0: page-cache, 1: direct>\n", argv[0]);
        return EXIT_SUCCESS;
    }

    size_order = atoi(argv[2]);

    use_direct = atoi(argv[3]);
    if (use_direct)
    {
        flags |= D_F_DIRECT;
        printf("Use D_F_DIRECT\n");
    }
    else
    {
        printf("NOT USING D_F_DIRECT\n");
    }
    
    total_size = ((size_t)1 << 30) * size_order;

    // Each block reads exactly 1 GPU page (2MB).
    num_tblocks = total_size / GPU_PAGE_SIZE;

    CUDA_CALL_SAFE(hipEventCreate(&start_event));
    CUDA_CALL_SAFE(hipEventCreate(&stop_event));

    gettimeofday(&tv_start, NULL);
    if (dragon_map(argv[1], total_size, flags, (void **)&g_buf) != D_OK)
        return EXIT_FAILURE;
    fprintf(stderr, "g_buf: %p\n", g_buf);
    gettimeofday(&tv_stop, NULL);

    map_time = time_diff(tv_start, tv_stop);

    CUDA_CALL_SAFE(hipEventRecord(start_event));
    kernel<<< num_tblocks, NUM_THREADS_PER_BLOCK >>>(g_buf);
    CUDA_CALL_SAFE(hipEventRecord(stop_event));

    CUDA_CALL_SAFE(hipEventSynchronize(stop_event));
    CUDA_CALL_SAFE(hipEventElapsedTime(&kernel_time, start_event, stop_event));

    CUDA_CALL_SAFE(hipDeviceSynchronize());

    gettimeofday(&tv_start, NULL);
    if (dragon_unmap((void *)g_buf) != D_OK)
        return EXIT_FAILURE;
    gettimeofday(&tv_stop, NULL);

    free_time = time_diff(tv_start, tv_stop);

    bw = ((double)total_size / (double)1024) / (double)kernel_time;

    printf("==> header: total_size (GB),kernel_time (ms),free_time (ms),map_time (ms),bw (MB/s)\n");
    printf("==> data: %d,%f,%f,%f,%f\n", size_order, kernel_time, free_time, map_time, bw);

    return EXIT_SUCCESS;
}

