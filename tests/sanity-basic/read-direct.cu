#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <dragon.h>

#define CUDA_CALL_SAFE(f) \
    do \
    {                                                        \
        hipError_t _cuda_error = f;                         \
        if (_cuda_error != hipSuccess)                      \
        {                                                    \
            fprintf(stderr,  \
                "%s, %d, CUDA ERROR: %s %s\n",  \
                __FILE__,   \
                __LINE__,   \
                hipGetErrorName(_cuda_error),  \
                hipGetErrorString(_cuda_error) \
            ); \
            abort(); \
            return EXIT_FAILURE; \
        } \
    } while (0)        

double time_diff(struct timeval tv_start, struct timeval tv_stop)
{
    return (double)(tv_stop.tv_sec - tv_start.tv_sec) * 1000.0 + (double)(tv_stop.tv_usec - tv_start.tv_usec) / 1000.0;
}

__device__ bool d_result = true;

__global__ void kernel(uint32_t* g_buf, int seed) 
{
    size_t idx = (size_t)blockDim.x * (size_t)blockIdx.x + (size_t)threadIdx.x;

    if (g_buf[idx] != (uint32_t)(idx * seed))
        d_result = false;
}

int main(int argc, char *argv[])
{
    uint32_t *g_buf;
    size_t num_tblocks;          
    size_t num_threads;          
    int size_order;
    size_t total_size;
    hipEvent_t start_event, stop_event;
    float kernel_time = 0;          // in ms
    double free_time = 0;          // in ms
    double map_time = 0;          // in ms
    bool h_result = true;
    int seed;

    struct timeval tv_start, tv_stop;

    if (argc != 5)
    {
        fprintf(stderr, "Usage: %s file size_in_GiB threads_per_block seed\n", argv[0]);
        return EXIT_SUCCESS;
    }

    size_order = atoi(argv[2]);
    num_threads = atoi(argv[3]);
    seed = atoi(argv[4]);
    
    //total_size = ((size_t)1 << 30) * size_order;
    total_size = (size_t)1000000000 * (size_t)size_order;
    num_tblocks = total_size / sizeof(uint32_t) / num_threads;

    CUDA_CALL_SAFE(hipEventCreate(&start_event));
    CUDA_CALL_SAFE(hipEventCreate(&stop_event));

    gettimeofday(&tv_start, NULL);
    if (dragon_map(argv[1], total_size, D_F_READ | D_F_DIRECT, (void **)&g_buf) != D_OK)
        return EXIT_FAILURE;
    fprintf(stderr, "g_buf: %p\n", g_buf);
    gettimeofday(&tv_stop, NULL);

    map_time = time_diff(tv_start, tv_stop);

    CUDA_CALL_SAFE(hipEventRecord(start_event));
    kernel<<< num_tblocks, num_threads >>>(g_buf, seed);
    CUDA_CALL_SAFE(hipEventRecord(stop_event));

    CUDA_CALL_SAFE(hipEventSynchronize(stop_event));
    CUDA_CALL_SAFE(hipEventElapsedTime(&kernel_time, start_event, stop_event));

    CUDA_CALL_SAFE(hipDeviceSynchronize());

    gettimeofday(&tv_start, NULL);
    if (dragon_unmap(g_buf) != D_OK)
        return EXIT_FAILURE;
    gettimeofday(&tv_stop, NULL);

    free_time = time_diff(tv_start, tv_stop);

    CUDA_CALL_SAFE(hipMemcpyFromSymbol(&h_result, HIP_SYMBOL(d_result), sizeof(d_result), 0, hipMemcpyDeviceToHost));

    printf("==> header: kernel_time (ms),free_time (ms),map_time (ms)\n");
    printf("==> data: %f,%f,%f\n", kernel_time, free_time, map_time);
    printf("==> Data validation: %s\n", h_result ? "Pass" : "Fail");

    if (h_result)
        return EXIT_SUCCESS;
    else
        return EXIT_FAILURE;
}

